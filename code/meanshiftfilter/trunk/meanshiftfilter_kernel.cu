#include "hip/hip_runtime.h"
#ifndef _MSFILTER_KERNEL_H_
#define _MSFILTER_KERNEL_H_

#include <stdio.h>
#include <cutil_inline.h>
#include "meanshiftfilter_common.h"


#define EPSILON 0.01f
#define LIMIT 100.0f

// declare texture reference for 2D float texture
texture<float4, 2, hipReadModeElementType> tex;


__global__ void meanshiftfilter(float4* d_src, float4* d_dst, 
		unsigned int width, unsigned int height,
		float sigmaS, float sigmaR,
		float rsigmaS, float rsigmaR, unsigned int limit)

{
	// NOTE: iteration count is for speed up purposes only - it
	//       does not have any theoretical importance
	int iter = 0;

	float x, y;	
	float diff0, diff1;
	float dx, dy, dl, du, dv;

	volatile float mvAbs;
	float wsum;

	// Traverse each data point applying mean shift
	// to each data point
	float yk[5];
	float Mh[5];

	int ix = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int iy = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;

	// Assign window center (window centers are
	// initialized by createLattice to be the point
	// data[i])	
	float4 luv = tex2D(tex, ix, iy); 	// float4 luv = d_src[i];
	
	yk[0] = ix;
	yk[1] = iy;
	yk[2] = luv.x;
	yk[3] = luv.y;
	yk[4] = luv.z;

	// Initialize mean shift vector
	Mh[0] = 0.0f;
	Mh[1] = 0.0f;
	Mh[2] = 0.0f;
	Mh[3] = 0.0f;
	Mh[4] = 0.0f;


	// Keep shifting window center until the magnitude squared of the
	// mean shift vector calculated at the window center location is
	// under a specified threshold (Epsilon)

	volatile float limitcycle[8] = 
	{ 
		12345678.0f,
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f 
	}; // Period-4 limit cycle detection

	do {
		// Shift window location
		yk[0] += Mh[0];
		yk[1] += Mh[1];
		yk[2] += Mh[2];
		yk[3] += Mh[3];
		yk[4] += Mh[4];

		// Calculate the mean shift vector at the new
		// window location using lattice

		// Initialize mean shift vector
		Mh[0] = 0.0f;
		Mh[1] = 0.0f;
		Mh[2] = 0.0f;
		Mh[3] = 0.0f;
		Mh[4] = 0.0f;

		// Initialize wsum
		wsum = 0.0f;

		// Perform lattice search summing
		// all the points that lie within the search
		// window defined using the kernel specified
		// by uniformKernel


		//Define bounds of lattice...
		//the lattice is a 2dimensional subspace whose
		//search window bandwidth is specified by sigmaS:

		int lX = yk[0] - sigmaS;
		int lY = yk[1] - sigmaS;
		int uX = yk[0] + sigmaS;
		int uY = yk[1] + sigmaS;


		lX = fmaxf(0.0f, lX);
		lY = fmaxf(0.0f, lY);
		uX = fminf(uX, width - 1);
		uY = fminf(uY, height - 1);

		
		//Perform search using lattice
		//Iterate once through a window of size sigmaS
		for(y = lY; y <= uY; y += 1) {
			for(x = lX; x <= uX; x += 1) {
			
				diff0 = 0.0f;

				//Determine if inside search window
				//Calculate distance squared of sub-space s	

				dx = (x - yk[0]) * rsigmaS;
				dy = (y - yk[1]) * rsigmaS;

				diff0 = dx * dx;
				diff0 += dy * dy;


				if (diff0 >= 1.0f) continue;	
				
				luv = tex2D(tex, x, y); 
				
				diff1 = 0.0f;
				
				dl = (luv.x - yk[2]) * rsigmaR;               
				du = (luv.y - yk[3]) * rsigmaR;               
				dv = (luv.z - yk[4]) * rsigmaR;               


				diff1 = dl * dl;

				if((yk[2] > 80.0f)) 
					diff1 += 3.0f * dl * dl;

				diff1 += du * du;
				diff1 += dv * dv;


				if (diff1 >= 1.0f) continue;


				// If its inside search window perform sum and count
				// For a uniform kernel weight == 1 for all feature points
				// considered point is within sphere => accumulate to mean
				Mh[0] += x;
				Mh[1] += y;
				Mh[2] += luv.x;
				Mh[3] += luv.y;
				Mh[4] += luv.z;
				wsum += 1.0f; //weight

			}

		}
		// When using uniformKernel wsum is always > 0 .. since weight == 1 and 
		// wsum += weight. @see uniformLSearch for details ...

		// determine the new center and the magnitude of the meanshift vector
		// meanshiftVector = newCenter - center;
		wsum = 1.0f/wsum; 
		
		Mh[0] = Mh[0] * wsum - yk[0];
		Mh[1] = Mh[1] * wsum - yk[1];
		Mh[2] = Mh[2] * wsum - yk[2];
		Mh[3] = Mh[3] * wsum - yk[3];
		Mh[4] = Mh[4] * wsum - yk[4];



		// Calculate its magnitude squared
		mvAbs = 0;
		mvAbs += Mh[0] * Mh[0];
		mvAbs += Mh[1] * Mh[1];
		mvAbs += Mh[2] * Mh[2];
		mvAbs += Mh[3] * Mh[3];
		mvAbs += Mh[4] * Mh[4];

		
		
		if (mvAbs == limitcycle[0] || 
		    mvAbs == limitcycle[1] || 
		    mvAbs == limitcycle[2] || 
		    mvAbs == limitcycle[3] ||
		    mvAbs == limitcycle[4] ||
		    mvAbs == limitcycle[5] ||
		    mvAbs == limitcycle[6] ||
		    mvAbs == limitcycle[7]) 
		{
			break;
				
		}

		limitcycle[0] = limitcycle[1];
		limitcycle[1] = limitcycle[2];
		limitcycle[2] = limitcycle[3];
		limitcycle[3] = limitcycle[4];
		limitcycle[4] = limitcycle[5];
		limitcycle[5] = limitcycle[6];
		limitcycle[6] = limitcycle[7];
		limitcycle[7] = mvAbs;
		
		
		
		// Increment iteration count
		iter++;
		
			
	} while((mvAbs >= EPSILON) && (iter < limit));


	// Shift window location
	yk[0] += Mh[0];
	yk[1] += Mh[1];
	yk[2] += Mh[2];
	yk[3] += Mh[3];
	yk[4] += Mh[4];


	luv = make_float4(yk[2], yk[3], yk[4], 0.0f);

	// store result into global memory
	int i = ix + iy * width;
	
	//printf("%d %d \n", i , iter);
	
	//__syncthreads();
	d_dst[i] = luv;

	return;
}



extern "C" void initTexture(int width, int height, void *h_flt)
{
	hipArray* d_array;
	int size = width * height * sizeof(float4);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4> ();

	cutilSafeCall(hipMallocArray(&d_array, &channelDesc, width, height )); 
	cutilSafeCall(hipMemcpyToArray(d_array, 0, 0, h_flt, size, hipMemcpyHostToDevice));

	// set texture parameters
	//    tex.addressMode[0] = hipAddressModeWrap;
	//    tex.addressMode[1] = hipAddressModeWrap;
	//    tex.filterMode = hipFilterModeLinear;
	tex.normalized = 0;	// access without normalized texture coordinates
				// [0, width -1] [0, height - 1]
	// bind the array to the texture
	cutilSafeCall(hipBindTextureToArray(tex, d_array, channelDesc));
}


extern "C" void meanShiftFilter(dim3 grid, dim3 threads, float4* d_src, float4* d_dst,
		unsigned int width, unsigned int height,
		float sigmaS, float sigmaR,
		float rsigmaS, float rsigmaR, unsigned int limit)
{
	meanshiftfilter<<< grid, threads>>>(d_src, d_dst, width, height, sigmaS, sigmaR, rsigmaS, rsigmaR, limit);
}


#endif // #ifndef _MSFILTER_KERNEL_H_
