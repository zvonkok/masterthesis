#include "hip/hip_runtime.h"
#ifndef _MSFILTER_KERNEL_H_
#define _MSFILTER_KERNEL_H_

#include <stdio.h>
#include <cutil_inline.h>
#include "meanshiftfilter_common.h"

// declare texture reference for 2D float texture
texture<float4, 2, hipReadModeElementType> tex;

__global__ void meanshiftfilter(
	float4* d_src, float4* d_dst, 
	float width, float height,
	float sigmaS, float sigmaR,
	float rsigmaS, float rsigmaR)
{

	// NOTE: iteration count is for speed up purposes only - it
	//       does not have any theoretical importance
	float iter = 0;
	float wsum;

	
	// Traverse each data point applying mean shift
	// to each data point
	float yk[5];
	float Mh[5];

	float ix = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	float iy = __mul24(blockIdx.y, blockDim.y) + threadIdx.y;
	
	// Assign window center (window centers are
	// initialized by createLattice to be the point
	// data[i])	
	float4 luv = tex2D(tex, ix, iy); 	// float4 luv = d_src[i];
	
	yk[0] = ix;
	yk[1] = iy;
	yk[2] = luv.x;
	yk[3] = luv.y;
	yk[4] = luv.z;

	// Initialize mean shift vector
	Mh[0] = 0.0f;
	Mh[1] = 0.0f;
	Mh[2] = 0.0f;
	Mh[3] = 0.0f;
	Mh[4] = 0.0f;


	// Keep shifting window center until the magnitude squared of the
	// mean shift vector calculated at the window center location is
	// under a specified threshold (Epsilon)

	float limitcycle[8] = 
	{ 
		12345678.0f,
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f, 
		12345678.0f 
	}; // Period-8 limit cycle detection

	
	float mag;  // magnitude squared
	
	do {
		// Shift window location
		yk[0] += Mh[0];
		yk[1] += Mh[1];
		yk[2] += Mh[2];
		yk[3] += Mh[3];
		yk[4] += Mh[4];


		// Calculate the mean shift vector at the new
		// window location using lattice

		// Initialize mean shift vector
		Mh[0] = 0.0f;
		Mh[1] = 0.0f;
		Mh[2] = 0.0f;
		Mh[3] = 0.0f;
		Mh[4] = 0.0f;

		// Initialize wsum
		wsum = 0.0f;

		// Perform lattice search summing
		// all the points that lie within the search
		// window defined using the kernel specified
		// by uniformKernel


		//Define bounds of lattice...
		//the lattice is a 2dimensional subspace whose
		//search window bandwidth is specified by sigmaS:

		float lX = (int)yk[0] - sigmaS;
		float lY = (int)yk[1] - sigmaS;
		float uX = yk[0] + sigmaS;
		float uY = yk[1] + sigmaS;

		lX = fmaxf(0.0f, lX);
		lY = fmaxf(0.0f, lY);
		uX = fminf(uX, width - 1.0f);
		uY = fminf(uY, height - 1.0f);

				
		float x, y;	
		
		//Perform search using lattice
		//Iterate once through a window of size sigmaS
		for(y = lY; y <= uY; y += 1) {
			for(x = lX; x <= uX; x += 1) {

				//Determine if inside search window
				//Calculate distance squared of sub-space s	
				float diff0 = 0.0f;

				//dx = (x - yk[0]) * rsigmaS;
				//dy = (y - yk[1]) * rsigmaS;
				
				float dx_0 = x - yk[0];
				float dy_0 = y - yk[1];
				
				float dx = dx_0 * rsigmaS;
				float dy = dy_0 * rsigmaS;

				float diff0_0 = dx * dx;
				float diff0_1 = dy * dy;
				
				diff0 = diff0_0 + diff0_1;

				if (diff0 >= 1.0f) continue;
				
				luv = tex2D(tex, x, y); 
				
				float diff1 = 0.0f;
				
				//dl = (luv.x - yk[2]) * rsigmaR;               
				//du = (luv.y - yk[3]) * rsigmaR;               
				//dv = (luv.z - yk[4]) * rsigmaR;               
				float dl_0 = luv.x - yk[2];               
				float du_0 = luv.y - yk[3];               
				float dv_0 = luv.z - yk[4];
				
				float dl = dl_0 * rsigmaR; 
				float du = du_0 * rsigmaR;
				float dv = dv_0 * rsigmaR;
				
					
				float diff1_0 = dl * dl;
				float diff1_1 = du * du;
				float diff1_2 = dv * dv;
				diff1 = diff1_0 + diff1_1 + diff1_2;
			
				
				if((yk[2] > 80.0f)) { 
					diff1 += 3.0f * dl * dl;
				}
			
				if (diff1 >= 1.0f) continue;
				

				// If its inside search window perform sum and count
				// For a uniform kernel weight == 1 for all feature points
				// considered point is within sphere => accumulate to mean
				Mh[0] += x;
				Mh[1] += y;
				Mh[2] += luv.x;
				Mh[3] += luv.y;
				Mh[4] += luv.z;
				wsum += 1.0f; //weight

			}

		}
		// When using uniform kernel wsum is always > 0 .. since weight == 1 and 
		// wsum += weight. 
		// determine the new center and the magnitude of the meanshift vector
		// meanshiftVector = newCenter - center;
		wsum = 1.0f/wsum; 
		
		//Mh[0] = Mh[0] * wsum - yk[0];
		//Mh[1] = Mh[1] * wsum - yk[1];
		//Mh[2] = Mh[2] * wsum - yk[2];
		//Mh[3] = Mh[3] * wsum - yk[3];
		//Mh[4] = Mh[4] * wsum - yk[4];
		
		float ms_0 = Mh[0] * wsum;
		float ms_1 = Mh[1] * wsum;
		float ms_2 = Mh[2] * wsum;
		float ms_3 = Mh[3] * wsum;
		float ms_4 = Mh[4] * wsum;
		
		Mh[0] = ms_0 - yk[0];
		Mh[1] = ms_1 - yk[1];
		Mh[2] = ms_2 - yk[2];
		Mh[3] = ms_3 - yk[3];
		Mh[4] = ms_4 - yk[4];
		



		// Calculate its magnitude squared
		mag = 0;
		
		float mag_0 = Mh[0] * Mh[0];
		float mag_1 = Mh[1] * Mh[1];
		float mag_2 = Mh[2] * Mh[2];
		float mag_3 = Mh[3] * Mh[3];
		float mag_4 = Mh[4] * Mh[4];
		mag = mag_0 + mag_1 + mag_2 + mag_3 + mag_4; 

		
		// Usually you don't do float == float but in this case
		// it is completely safe as we have limit cycles where the 
		// values after some iterations are equal, the same
		if (mag == limitcycle[0] || 
		    mag == limitcycle[1] || 
		    mag == limitcycle[2] || 
		    mag == limitcycle[3] ||
		    mag == limitcycle[4] ||
		    mag == limitcycle[5] ||
		    mag == limitcycle[6] ||
		    mag == limitcycle[7]) 
		{
			break;
				
		}

		limitcycle[0] = limitcycle[1];
		limitcycle[1] = limitcycle[2];
		limitcycle[2] = limitcycle[3];
		limitcycle[3] = limitcycle[4];
		limitcycle[4] = limitcycle[5];
		limitcycle[5] = limitcycle[6];
		limitcycle[6] = limitcycle[7];
		limitcycle[7] = mag;
		
				
		// Increment iteration count
		iter += 1;
			
	} while((mag >= EPSILON) && (iter < LIMIT));


	// Shift window location
	yk[0] += Mh[0];
	yk[1] += Mh[1];
	yk[2] += Mh[2];
	yk[3] += Mh[3];
	yk[4] += Mh[4];


	luv = make_float4(yk[2], yk[3], yk[4], 0.0f);

	// store result into global memory
	float i = ix + iy * width;
	d_dst[(int)i] = luv;

	
	return;
}



extern "C" void initTexture(int width, int height, void *h_flt)
{
	hipArray* d_array;
	int size = width * height * sizeof(float4);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4> ();

	cutilSafeCall(hipMallocArray(&d_array, &channelDesc, width, height )); 
	cutilSafeCall(hipMemcpyToArray(d_array, 0, 0, h_flt, size, hipMemcpyHostToDevice));

	// set texture parameters
	//    tex.addressMode[0] = hipAddressModeWrap;
	//    tex.addressMode[1] = hipAddressModeWrap;
	//    tex.filterMode = hipFilterModeLinear;
	tex.normalized = 0;	// access without normalized texture coordinates
				// [0, width -1] [0, height - 1]
	// bind the array to the texture
	cutilSafeCall(hipBindTextureToArray(tex, d_array, channelDesc));
}


extern "C" void meanShiftFilter(dim3 grid, dim3 threads, float4* d_src, float4* d_dst,
		float width, float height,
		float sigmaS, float sigmaR,
		float rsigmaS, float rsigmaR)
{
	meanshiftfilter<<< grid, threads>>>(d_src, d_dst, width, height, sigmaS, sigmaR, rsigmaS, rsigmaR);
}


#endif // #ifndef _MSFILTER_KERNEL_H_
