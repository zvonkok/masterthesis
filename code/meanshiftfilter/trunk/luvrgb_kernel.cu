
#include <hip/hip_runtime.h>


const float Yn = 1.00000f;
const double Un_prime	= 0.19784977571475;
const double Vn_prime	= 0.46834507665248;



//define inline rounding function...
__device__ int my_round(float in_x)
{
	if (in_x < 0)
		return (int)(in_x - 0.5);
	else
		return (int)(in_x + 0.5);
}

// convert floating point rgba color to 32-bit integer
__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}


__global__ void luvtorgb(float4 *d_luv, unsigned int *d_rgb, unsigned int width)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int i = ix + iy * width;
	
	float4 luv = d_luv[i];
	
	
	
	float RGB[3][3] = {
		{  3.2405f, -1.5371f, -0.4985f },
		{ -0.9693f,  1.8760f,  0.0416f },
		{  0.0556f, -0.2040f,  1.0573f }	
	};

	//declare variables...
	int	  r, g, b;
	float x, y, z, u_prime, v_prime;
	
	//perform conversion
	if(luv.x < 0.1)
		r = g = b = 0;
	else
	{
		//convert luv to xyz...
		if(luv.x < 8.0)
			y	= Yn * luv.x / 903.3;
		else
		{
			y = (luv.x + 16.0) / 116.0;
			y *= Yn * y * y;
		}
		
		u_prime	= luv.y / (13 * luv.x) + Un_prime;
		v_prime	= luv.z / (13 * luv.x) + Vn_prime;
		
		x = 9 * u_prime * y / (4 * v_prime);
		z = (12 - 3 * u_prime - 20 * v_prime) * y / (4 * v_prime);
		
		//convert xyz to rgb...
		//[r, g, b] = RGB*[x, y, z]*255.0
		r = my_round((RGB[0][0]*x + RGB[0][1]*y + RGB[0][2]*z)*255.0);
		g = my_round((RGB[1][0]*x + RGB[1][1]*y + RGB[1][2]*z)*255.0);
		b = my_round((RGB[2][0]*x + RGB[2][1]*y + RGB[2][2]*z)*255.0);
		
		//check bounds...
		if(r < 0)	r = 0; if(r > 255)	r = 255;
		if(g < 0)	g = 0; if(g > 255)	g = 255;
		if(b < 0)	b = 0; if(b > 255)	b = 255;
		
	}
	
	//assign rgb values to rgb vector rgb
	//rgb[0] = r;
	//rgb[1] = g;
	//rgb[2] = b;
	
	//float4 rgba = { rgb[0], rgb[1], rgb[2], 0.0f };
	
	//d_rgb[i] = rgbaFloatToInt(rgba);
	//done.
	
	((unsigned char*)&d_rgb[i])[0] = r;
	((unsigned char*)&d_rgb[i])[1] = g;
	((unsigned char*)&d_rgb[i])[2] = b;
	((unsigned char*)&d_rgb[i])[3] = 0;
	
	return;
	
}

extern "C" void luvToRgb(dim3 grid, dim3 threads, float4* d_luv, 
	unsigned int* d_rgb, unsigned int width)
{
	luvtorgb<<< grid, threads >>>(d_luv, d_rgb, width);
}


